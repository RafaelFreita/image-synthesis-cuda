#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iterator>
#include <limits>
#include <numeric>
#include <string>

#include <stb_image.h>
#include "stb_image_write.h"

#include "Timer.hpp"
#include "cxxopts.hpp"

struct Image {
	int width, height, channelsQtd;
	uint8_t* data;

	bool loadedImage = false;
	bool cleaned = false;

	bool Load(char const* path) {
		data = stbi_load(path, &width, &height, &channelsQtd, 0);
		if (!data) {
			FreeImage();
			return false;
		}

		loadedImage = true;
		return true;
	}

	void FreeImage() {
		if (cleaned || !loadedImage) return;
		cleaned = true;
		stbi_image_free(data);
	}

	int GetDataSize() {
		return width * height * channelsQtd;
	}

	void AllocateDataArray() {
		data = new uint8_t[width * height * channelsQtd];
	}

	void WriteImage(char const* path) {
		stbi_write_png(path, width, height, channelsQtd, data, width * channelsQtd);
	}

	~Image() {
		if (!cleaned) FreeImage();
	}

};

#pragma region Helpers

float distanceBetweenColors(uint8_t r1, uint8_t g1, uint8_t b1, uint8_t r2, uint8_t g2, uint8_t b2) {
	int16_t r = r2 - r1;
	int16_t g = g2 - g1;
	int16_t b = b2 - b1;
	float d = r * r + g * g + b * b;
	return sqrt(d);
}

__device__ float distanceBetweenColorsCuda(uint8_t r1, uint8_t g1, uint8_t b1, uint8_t r2, uint8_t g2, uint8_t b2) {
	int16_t r = r2 - r1;
	int16_t g = g2 - g1;
	int16_t b = b2 - b1;
	float d = r * r + g * g + b * b;
	return sqrt(d);
}

uint16_t wrapValue(uint16_t a, uint16_t b) {
	if (a < 0) return b + a;
	else if (a >= b) return abs(b - a);
	return a;
}

double computeAverage(std::vector<double>& v)
{
	size_t n = v.size();
	if (n == 0)
		return 0.0;

	return std::accumulate(v.begin(), v.end(), 0.0) / n;
}

double computeMedian(std::vector<double>& v)
{
	size_t n = v.size() / 2;
	std::nth_element(v.begin(), v.begin() + n, v.end()); // Like std::sort, but only sorts until nth element
	return v[n];
}

#pragma endregion

#pragma region CPU

unsigned char* synthetiseImage(Image& sampleImage, Image& preSynImage, uint8_t neighborhoodSize) {

	if (neighborhoodSize % 2 == 0) {
		printf("ERROR neighborhoodSize for image synthesis must be odd!\n");
		return nullptr;
	}

	int arrSize = preSynImage.width * preSynImage.height * 3;
	unsigned char* synthetizedImage = new unsigned char[arrSize];

	for (int i = 0; i < arrSize; i++)
	{
		synthetizedImage[i] = preSynImage.data[i];
	}

	int neighborSynX = 0, neighborSynY = 0;
	int neighborSampleX = 0, neighborSampleY = 0;

	uint16_t winnerPixelPosX, winnerPixelPosY;
	float winnerPontuation;
	float pontuation = 0;

	uint8_t neighborhoodHalf = (neighborhoodSize / 2);

	// For every pixel
	for (int y = 0; y < preSynImage.height; y++)
	{
		for (int x = 0; x < preSynImage.width; x++)
		{

			winnerPontuation = std::numeric_limits<float>::max();

			// For every pixel in sample image
			for (int sY = 0; sY < sampleImage.height; sY++)
			{
				for (int sX = 0; sX < sampleImage.width; sX++)
				{

					pontuation = 0.0f;

					// Check neighbors - From upper corner to the pixel itself
					for (int nY = -neighborhoodHalf; nY <= 0; nY++)
					{

						neighborSynY = y + nY;
						if (neighborSynY < 0) {
							neighborSynY = preSynImage.height + neighborSynY;
						}
						else if (neighborSynY >= preSynImage.height) {
							neighborSynY = abs(preSynImage.height - neighborSynY);
						}

						neighborSampleY = sY + nY;
						if (neighborSampleY < 0) {
							neighborSampleY = sampleImage.height + neighborSampleY;
						}
						else if (neighborSampleY >= sampleImage.height) {
							neighborSampleY = abs(sampleImage.height - neighborSampleY);
						}

						for (int nX = -neighborhoodHalf; nX <= neighborhoodHalf; nX++)
						{
							if (nY == 0 && nX > 0) { // Last line going only until the proper pixel
								break;
							}

							neighborSynX = x + nX;
							if (neighborSynX < 0) {
								neighborSynX = preSynImage.width + neighborSynX;
							}
							else if (neighborSynX >= preSynImage.width) {
								neighborSynX = abs(preSynImage.width - neighborSynX);
							}

							neighborSampleX = sX + nX;
							if (neighborSampleX < 0) {
								neighborSampleX = sampleImage.width + neighborSampleX;
							}
							else if (neighborSampleX >= sampleImage.width) {
								neighborSampleX = abs(sampleImage.width - neighborSampleX);
							}

							// Compare pixels from sample with preSyn
							// 0 - R // 1 - G // 2 - B

							int newImagePos = neighborSynX * preSynImage.channelsQtd + (neighborSynY * preSynImage.width * preSynImage.channelsQtd);
							int tempPos = neighborSampleX * sampleImage.channelsQtd + (neighborSampleY * sampleImage.width * sampleImage.channelsQtd);

							float dist = distanceBetweenColors(
								synthetizedImage[newImagePos + 0],
								synthetizedImage[newImagePos + 1],
								synthetizedImage[newImagePos + 2],

								sampleImage.data[tempPos + 0],
								sampleImage.data[tempPos + 1],
								sampleImage.data[tempPos + 2]
							);

							pontuation += dist;
						}
					}
					// End of neighbors

					// Assign new winner if pontution was smaller
					if (pontuation < winnerPontuation) {
						winnerPixelPosX = sX;
						winnerPixelPosY = sY;
						winnerPontuation = pontuation;
					}

				}
			}
			// End of sample check

			int pixelPos = x * preSynImage.channelsQtd + (y * preSynImage.width * preSynImage.channelsQtd);
			int winerPixelPos = winnerPixelPosX * sampleImage.channelsQtd + (winnerPixelPosY * sampleImage.width * sampleImage.channelsQtd);
			synthetizedImage[pixelPos + 0] = sampleImage.data[winerPixelPos + 0];
			synthetizedImage[pixelPos + 1] = sampleImage.data[winerPixelPos + 1];
			synthetizedImage[pixelPos + 2] = sampleImage.data[winerPixelPos + 2];

		}
	}

	return synthetizedImage;
}

#pragma endregion

#pragma region GPU
hipError_t synthetiseImageCuda(
	uint16_t sampleWidth, uint16_t sampleHeight, uint16_t sampleChannels, uint8_t* sampleData,
	uint16_t preSynWidth, uint16_t preSynHeight, uint16_t preSynChannels, uint8_t* preSynData,
	uint8_t neighborhood, uint8_t* synthetizedData, uint8_t threadsPerBlock = 16U);

__global__ void synthetiseImageKernel(
	int16_t sampleWidth, int16_t sampleHeight, int16_t sampleChannels, uint8_t* sampleData,
	int16_t preSynWidth, int16_t preSynHeight, int16_t preSynChannels, uint8_t* synthetizedData,
	int16_t neighborhoodHalf, float* pontuationArray, int16_t x, int16_t y) {

	/*int arrSize = preSynWidth * preSynHeight * 3;
	unsigned char* newImage = new unsigned char[arrSize];

	for (int i = 0; i < arrSize; i++)
	{
		newImage[i] = preSynData[i];
	}*/

	int32_t neighborSynX = 0, neighborSynY = 0;
	int32_t neighborSampleX = 0, neighborSampleY = 0;

	float pontuation = 0;

	int16_t sX = threadIdx.x + blockIdx.x * blockDim.x;
	int16_t sY = threadIdx.y + blockIdx.y * blockDim.y;

	if (sX >= sampleWidth || sY >= sampleHeight) return;

	for (int nY = -neighborhoodHalf; nY <= 0; nY++)
	{

		neighborSynY = y + nY;
		if (neighborSynY < 0) neighborSynY = preSynHeight + neighborSynY;
		else if (neighborSynY >= preSynHeight) neighborSynY = abs(preSynHeight - neighborSynY);

		neighborSampleY = sY + nY;
		if (neighborSampleY < 0) neighborSampleY = sampleHeight + neighborSampleY;
		else if (neighborSampleY >= sampleHeight) neighborSampleY = abs(sampleHeight - neighborSampleY);

		for (int nX = -neighborhoodHalf; nX <= neighborhoodHalf; nX++)
		{
			if (nY == 0 && nX > 0) { // Last line going only until the proper pixel
				break;
			}

			neighborSynX = x + nX;
			if (neighborSynX < 0) neighborSynX = preSynWidth + neighborSynX;
			else if (neighborSynX >= preSynWidth) neighborSynX = abs(preSynWidth - neighborSynX);

			neighborSampleX = sX + nX;
			if (neighborSampleX < 0) neighborSampleX = sampleWidth + neighborSampleX;
			else if (neighborSampleX >= sampleWidth) neighborSampleX = abs(sampleWidth - neighborSampleX);

			// Compare pixels from sample with preSyn
			// 0 - R // 1 - G // 2 - B

			int newImagePos = neighborSynX * preSynChannels + (neighborSynY * preSynWidth * preSynChannels);
			int tempPos = neighborSampleX * sampleChannels + (neighborSampleY * sampleWidth * sampleChannels);

			float dist = distanceBetweenColorsCuda(
				synthetizedData[newImagePos + 0],
				synthetizedData[newImagePos + 1],
				synthetizedData[newImagePos + 2],

				sampleData[tempPos + 0],
				sampleData[tempPos + 1],
				sampleData[tempPos + 2]
			);

			pontuation += dist;
		}
	}

	int pontuationIndex = sX + sY * sampleWidth;
	pontuationArray[pontuationIndex] = pontuation;

}

#pragma endregion

int main(int argc, char* argv[])
{

#pragma region Command Line Options
	cxxopts::Options options("Cuda Image Synthesis", "Synthetyse an image using a sample and a pre-synthesis image with Cuda. \nRafael de Freitas, 2020\n");

	options.add_options()
		("h,help", "Print usage")
		("c,cpu", "Run on CPU", cxxopts::value<bool>()->default_value("false"))
		("g,gpu", "Run on GPU", cxxopts::value<bool>()->default_value("true"))
		("t,threads", "How many threads per block", cxxopts::value<int>()->default_value("16"))
		("n,neighborhood", "Neighborhood size - Must be odd", cxxopts::value<int>()->default_value("5"))
		("s,sample", "Sample image path", cxxopts::value<std::string>()->default_value(""))
		("p,presyn", "Pre-synthesis image path", cxxopts::value<std::string>()->default_value(""))
		("r,result", "Result image path WITHOUT EXTENSION", cxxopts::value<std::string>()->default_value(""))
		("i,itt", "How many tests to run", cxxopts::value<int>()->default_value("1"))
		;

	auto optionsResult = options.parse(argc, argv);

	if (optionsResult.count("help"))
	{
		std::cout << options.help() << std::endl;
		return 0;
	}

	const bool runCPU = optionsResult["cpu"].as<bool>();
	const bool runGPU = optionsResult["gpu"].as<bool>();

	const std::string pathSample = optionsResult["sample"].as<std::string>();
	const std::string pathPresyn = optionsResult["presyn"].as<std::string>();
	const std::string pathResult = optionsResult["result"].as<std::string>();

	const int neighborhoodSize = optionsResult["neighborhood"].as<int>();
	const int threadsPerBlock = optionsResult["threads"].as<int>();

	const int testIterations = optionsResult["itt"].as<int>();

	//
	// Checking requirements
	//
	if (pathSample == "") {
		printf("ERROR Can't run without sample path! \n");
		return 1;
	}

	if (pathPresyn == "") {
		printf("ERROR Can't run without pre-synthesis path! \n");
		return 1;
	}

	if (pathResult == "") {
		printf("ERROR Can't run without result path! \n");
		return 1;
	}

	if (neighborhoodSize % 2 == 0) {
		printf("ERROR NeighborhoodSize must be odd!\n");
		return 1;
	}

#pragma endregion

	//
	// Loading images
	//
	stbi_set_flip_vertically_on_load(true);

	Image sampleImg;
	sampleImg.Load(pathSample.c_str());

	Image preSynImg;
	preSynImg.Load(pathPresyn.c_str());

	Timer timer = Timer();
	std::vector<double> execTimesCPU, execTimesGPU;

	if (runCPU) {
		unsigned char* synData;

		for (int i = 0; i < testIterations; i++) {
			timer.start();
			synData = synthetiseImage(sampleImg, preSynImg, neighborhoodSize);
			if (synData == nullptr) {
				system("pause");
				return 1;
			}
			timer.finish();
			execTimesCPU.push_back(timer.getElapsedTimeMs());
		}

		Image synthetizedImageCPU;
		synthetizedImageCPU.width = preSynImg.width;
		synthetizedImageCPU.height = preSynImg.height;
		synthetizedImageCPU.channelsQtd = 3;
		synthetizedImageCPU.data = synData;

		if(!runGPU)
			synthetizedImageCPU.WriteImage((pathResult + ".png").c_str());
		else
			synthetizedImageCPU.WriteImage((pathResult + "-CPU.png").c_str());

		printf("CPU Average Time (ms): %f\n", computeAverage(execTimesCPU));
		printf("CPU Median Time (ms): %f\n", computeMedian(execTimesCPU));
		printf("\n");
	}

	if (runGPU) {
		Image synthetizedImageGPU;
		synthetizedImageGPU.width = preSynImg.width;
		synthetizedImageGPU.height = preSynImg.height;
		synthetizedImageGPU.channelsQtd = 3;
		synthetizedImageGPU.AllocateDataArray();

#pragma region Cuda calls
		for (int i = 0; i < testIterations; i++) {
			timer.start();
			hipError_t cudaStatus = synthetiseImageCuda(
				sampleImg.width, sampleImg.height, sampleImg.channelsQtd, sampleImg.data,
				preSynImg.width, preSynImg.height, preSynImg.channelsQtd, preSynImg.data,
				5, synthetizedImageGPU.data, threadsPerBlock
			);
			timer.finish();

			execTimesGPU.push_back(timer.getElapsedTimeMs());

			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "synthetizeCuda failed!\n");
				system("pause");
				return 1;
			}

			cudaStatus = hipDeviceReset();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceReset failed!\n");
				system("pause");
				return 1;
			}
		}

		if (!runCPU)
			synthetizedImageGPU.WriteImage((pathResult + ".png").c_str());
		else
			synthetizedImageGPU.WriteImage((pathResult + "-GPU.png").c_str());

		printf("GPU Average Time (ms): %f\n", computeAverage(execTimesGPU));
		printf("GPU Median Time (ms): %f\n", computeMedian(execTimesGPU));
		printf("\n");
#pragma endregion

	}

	//
	// Cleanup
	//
	sampleImg.FreeImage();
	preSynImg.FreeImage();

	return 0;
}

hipError_t synthetiseImageCuda(uint16_t sampleWidth, uint16_t sampleHeight, uint16_t sampleChannels, uint8_t* sampleData,
	uint16_t preSynWidth, uint16_t preSynHeight, uint16_t preSynChannels, uint8_t* preSynData,
	uint8_t neighborhood, uint8_t* synthetizedData, uint8_t threadsPerBlock) {

	const uint32_t preSynSize = preSynWidth * preSynHeight * preSynChannels;
	const size_t preSynSizeBytes = preSynSize * sizeof(uint8_t);
	const size_t sampleSizeBytes = sampleWidth * sampleHeight * sampleChannels * sizeof(uint8_t);

	uint8_t* cuda_sampleData = 0;
	uint8_t* cuda_synthetizedData = 0;

	const uint32_t pontuationsArraySize = sampleWidth * sampleHeight;
	float* pontuationsArray = new float[pontuationsArraySize];
	float* cuda_pontuationsArray = 0;

	for (int i = 0; i < preSynSize; i++)
	{
		synthetizedData[i] = preSynData[i];
	}

	uint8_t neighborhoodHalf = neighborhood / 2;
	hipError_t cudaStatus;

#pragma region Cuda Memory Allocation

	// Set Cuda device
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Alocate arrays in GPU
	cudaStatus = hipMalloc((void**)& cuda_synthetizedData, preSynSizeBytes);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& cuda_sampleData, sampleSizeBytes);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& cuda_pontuationsArray, sizeof(float) * pontuationsArraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy sample data
	cudaStatus = hipMemcpy(cuda_sampleData, sampleData, sampleSizeBytes, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy synthesis data
	cudaStatus = hipMemcpy(cuda_synthetizedData, synthetizedData, preSynSizeBytes, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

#pragma endregion

	dim3 threadsPerBlockDim(threadsPerBlock, threadsPerBlock);
	// Adding one block on each dimension for non-square images. The boundary check is inside the kernel.
	dim3 numBlocks(sampleWidth / threadsPerBlockDim.x + 1, sampleHeight / threadsPerBlockDim.y + 1);

	for (int y = 0; y < preSynHeight; y++)
	{
		for (int x = 0; x < preSynWidth; x++)
		{

			synthetiseImageKernel << < numBlocks, threadsPerBlockDim >> > (
				sampleWidth, sampleHeight, sampleChannels, cuda_sampleData,
				preSynWidth, preSynHeight, preSynChannels, cuda_synthetizedData,
				neighborhoodHalf, cuda_pontuationsArray, x, y);

			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "synthetiseImageKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}

			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching synthetiseImageKernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
				goto Error;
			}

			cudaStatus = hipMemcpy(pontuationsArray, cuda_pontuationsArray, pontuationsArraySize * sizeof(float), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy pontuationsArray failed!");
				goto Error;
			}

			int winnerPontuationItt = 0;
			float winnerPontuation = std::numeric_limits<float>::max();
			for (int pontuationItt = 0; pontuationItt < pontuationsArraySize; pontuationItt++) {
				if (pontuationsArray[pontuationItt] < winnerPontuation) {
					winnerPontuation = pontuationsArray[pontuationItt];
					winnerPontuationItt = pontuationItt;
				}
			}

			int winnerPixelPos = winnerPontuationItt * sampleChannels;

			int pixelPos = x * preSynChannels + (y * preSynWidth * preSynChannels);
			synthetizedData[pixelPos + 0] = sampleData[winnerPixelPos + 0];
			synthetizedData[pixelPos + 1] = sampleData[winnerPixelPos + 1];
			synthetizedData[pixelPos + 2] = sampleData[winnerPixelPos + 2];

			cudaStatus = hipMemcpy(cuda_synthetizedData, synthetizedData, preSynSizeBytes, hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				goto Error;
			}
		}
	}

	// Copy synthetized data back
	cudaStatus = hipMemcpy(synthetizedData, cuda_synthetizedData, preSynSizeBytes, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(cuda_synthetizedData);
	hipFree(cuda_sampleData);

	return cudaStatus;
}